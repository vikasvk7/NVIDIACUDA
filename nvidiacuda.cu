
#include <hip/hip_runtime.h>
#include <stdio.h>
//Compiler version gcc 6.3.0


__global__ void sum(int *a, int *b, int *c) 
{ 
	*c = *a + *b; 
    } 
int main(void) 
{ 
	int a, b, c; // host copies of a, b, c 
    int *d_a, *d_b, *d_c; // device copies of a, b, c
    int size = sizeof(int); 
    // Allocate space for device copies of a, b, c 
    hipMalloc((void **)&d_a, size); 
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size); 
    //Input values
    a = 10; b = 20; 
   
 
// Copy inputs to device
hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice); 
hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice); 
//Launching add() kernel on GPU
sum<<<1,1>>>(d_a, d_b, d_c); 
// Copy result back to host
hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost); // Cleanup 
hipFree(d_a); 
hipFree(d_b);
hipFree(d_c);
printf("%d",c); 
return 0; 
        
}

